#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <ctype.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <time.h>
#include <sys/time.h>

#include "Process.h"
#include "My_Matrix.h"
#include "Process.cpp"
#include "My_Matrix.cpp"



#include "kernels.cu"

#define IMG_DATA_OFFSET_POS 10
#define BITS_PER_PIXEL_POS 28


int main(int argc, char *argv[]) 
{
	int ii;
	int fd, fd1;
	char *fdata;
	char *alldata, *all;
	struct stat finfo;
	char * outputfname;
	int numPic = TRAIN_NUM;
	unsigned short *data_pos;
	int width, height, fileSize;
	char inputfname[8];


	double *T,*T1,*T_test,*m1;
	unsigned char *m;
	double *L, *b,*q,*c,*p_q,*projected_train,*projected_test,*eigenvector,*Euc_dist;  
    unsigned int *L1;
    double eps;
	double temp;  
    int i,j,flag,iteration,num_q;  

      		

	T = (double *)malloc(sizeof(double)*IMG_HEIGHT*IMG_WIDTH*TRAIN_NUM);
    T_test = (double*)malloc(sizeof(double)*IMG_HEIGHT*IMG_WIDTH*1);    
    m = (unsigned char *)malloc(sizeof(unsigned char)*IMG_HEIGHT*IMG_WIDTH);
	m1 = (double *)malloc(sizeof(double)*IMG_HEIGHT*IMG_WIDTH);
    L = (double *)malloc(sizeof(double)*TRAIN_NUM*TRAIN_NUM);   
	L1 = (unsigned int *)malloc(sizeof(unsigned int)*(numPic + 1)*numPic/2);    
    b = (double *)malloc(sizeof(double)*TRAIN_NUM);         
    q = (double *)malloc(sizeof(double)*TRAIN_NUM*TRAIN_NUM);
    c = (double *)malloc(sizeof(double)*TRAIN_NUM);             
	T1 = (double *)malloc(sizeof(double)*IMG_HEIGHT*IMG_WIDTH*TRAIN_NUM);




	if (argc < 2)
	{
		printf("USAGE: %s <picture filename>\n", argv[0]);
		exit(1);
	}

	outputfname = argv[1];

	printf("Match Process: Running...\n");

	for(int fla = 1; fla <= numPic*4; fla ++)
	{
		if(fla <= 9)
		{
			inputfname[0] = (char)(fla +'0');
			inputfname[1] = '.';
			inputfname[2] = 'b';
		    inputfname[3] = 'm';
			inputfname[4] = 'p';
			inputfname[5] = '\0';
		} else if(fla>9 && fla<=19){
			inputfname[0] = '1';
			inputfname[1] = (char)(fla - 10 +'0');
			inputfname[2] = '.';
			inputfname[3] = 'b';
		    inputfname[4] = 'm';
			inputfname[5] = 'p';
			inputfname[6] = '\0';
		} else if(fla>19 && fla<=29){
			inputfname[0] = '2';
			inputfname[1] = (char)(fla - 20 +'0');
		} else if(fla>29 && fla<=39){
			inputfname[0] = '3';
			inputfname[1] = (char)(fla - 30 +'0');
		} else if(fla>39 && fla<=49){
			inputfname[0] = '4';
			inputfname[1] = (char)(fla - 40 +'0');
		} else if(fla>49 && fla<=59){
			inputfname[0] = '5';
			inputfname[1] = (char)(fla - 50 +'0');
		} else if(fla>59 && fla<=69){
			inputfname[0] = '6';
			inputfname[1] = (char)(fla - 60 +'0');
		} else if(fla>69 && fla<=79){
			inputfname[0] = '7';
			inputfname[1] = (char)(fla - 70 +'0');
		} else if(fla>79 && fla<=89){
			inputfname[0] = '8';
			inputfname[1] = (char)(fla - 80 +'0');
		} else if(fla>89 && fla<=99){
			inputfname[0] = '9';
			inputfname[1] = (char)(fla - 90 +'0');
		} else {
			inputfname[0] = '1';
			inputfname[1] = '0';
			inputfname[2] = '0';
			inputfname[3] = '.';
			inputfname[4] = 'b';
			inputfname[5] = 'm';
			inputfname[6] = 'p';
			inputfname[7] = '\0';
		}
		fd = open(inputfname, O_RDONLY);
		fstat(fd, &finfo);


		if(fla == 1)
		{
			alldata = (char*) malloc(numPic * 4* finfo.st_size);
			all = (char*) malloc(numPic * finfo.st_size);
		}

		fdata = (char*) malloc(finfo.st_size);

		read (fd, fdata, finfo.st_size);
	
		data_pos = (unsigned short *)(&(fdata[IMG_DATA_OFFSET_POS]));
	

		width = *((int*)&fdata[18]);
		//printf("Width: %d\n", width);
		height = *((int*)&fdata[22]);
		//printf("Height: %d\n", height);

		fileSize = (int) finfo.st_size;	

		char* p = &(fdata[*data_pos]);

		memcpy(&alldata[0 + (fla - 1)* width * height * 3], p, width * height * 3);

	}
	
	dim3 grid_m(36, 1, 1);
	dim3 block_m(1000, 1, 1);

	dim3 grid(32, 1, 1);
	dim3 block(1024, 1, 1);
	
	unsigned char* d_input;
	hipMalloc((void**) &d_input, numPic *4* width * height * 3);
    hipMemcpy(d_input, alldata, numPic *4* width * height * 3, hipMemcpyHostToDevice);
	
	
	unsigned char* d_output;
	hipMalloc((void**) &d_output, numPic * width * height *3);
	hipMemset(d_output, 0, numPic * width * height*3);
	
	struct timeval start_tv, end_tv;
	time_t sec;
	time_t ms;
	time_t diff;
	gettimeofday(&start_tv, NULL);
	
	mean<<<grid_m, block_m >>>((uchar3*) d_input, (uchar3*) d_output, numPic);
	
	hipDeviceSynchronize();
	
	gettimeofday(&end_tv, NULL);
	sec = end_tv.tv_sec - start_tv.tv_sec;
	ms = end_tv.tv_usec - start_tv.tv_usec;

	diff = sec * 1000000 + ms;

	//printf("%10s:\t\t%fms\n", "Time elapsed", (double)((double)diff/1000.0));
	
	hipMemcpy(all, d_output, numPic * height * width*3, hipMemcpyDeviceToHost);
	hipFree(d_input);
	hipFree(d_output);
	
	
	

	unsigned char* d_inputPixels;
	hipMalloc((void**) &d_inputPixels, numPic * width * height * 3);
    hipMemcpy(d_inputPixels, all, numPic * width * height * 3, hipMemcpyHostToDevice);
	
	
	unsigned char* d_outputPixels;
	hipMalloc((void**) &d_outputPixels, numPic * width * height );
	hipMemset(d_outputPixels, 0, numPic * width * height);

	unsigned char* d_m;
	hipMalloc((void**) &d_m, width * height );
	hipMemset(d_m, 0, width * height);

	
	unsigned char* outputPixels = (unsigned char*) malloc(numPic * height * width );


	gettimeofday(&start_tv, NULL);


	rgb2gray_mean<<<grid, block ,numPic * 1024 * sizeof(unsigned char)>>>((uchar3*) d_inputPixels, (unsigned char*) d_outputPixels, numPic, (unsigned char*) d_m);
	
	hipDeviceSynchronize();

	gettimeofday(&end_tv, NULL);
	sec = end_tv.tv_sec - start_tv.tv_sec;
	ms = end_tv.tv_usec - start_tv.tv_usec;

	diff = sec * 1000000 + ms;

	//printf("%10s:\t\t%fms\n", "Time elapsed", (double)((double)diff/1000.0));

	hipMemcpy(outputPixels, d_outputPixels, numPic * height * width, hipMemcpyDeviceToHost);
	//hipMemcpy(T, d_outputPixels, numPic * height * width, hipMemcpyDeviceToHost);
	hipMemcpy(m, d_m, 36000, hipMemcpyDeviceToHost);


	memcpy(&(fdata[*data_pos]), &outputPixels[(numPic- 1)*height*width],  height * width);
	

	hipFree(d_inputPixels);
	hipFree(d_outputPixels);
	hipFree(d_m);
	free(alldata);

	hipDeviceReset();




	eps = 0.000001;  
    memset(L,0,sizeof(double)*TRAIN_NUM*TRAIN_NUM);  

    dim3 grid_L(36,1,1);
	dim3 block_L(1000,1,1);
	unsigned int* d_L;
	hipMalloc((void**) &d_L, ((numPic + 1)*numPic)/2);
    hipMemset(d_L, 0, ((numPic + 1)*numPic)/2);

	unsigned int* d_tem;
	hipMalloc((void**) &d_tem, 36);
    hipMemset(d_tem, 0, 36);

	//unsigned char* d_outputPixels;
	hipMalloc((void**) &d_outputPixels, numPic * width * height );
	hipMemcpy(d_outputPixels, outputPixels, numPic * width * height, hipMemcpyHostToDevice);


	gettimeofday(&start_tv, NULL);

	calc_covariance_matrix_1<<<grid_L, block_L>>>((unsigned char*) d_outputPixels, numPic, (unsigned int*) d_L, (unsigned int*)d_tem);
	hipDeviceSynchronize();

	gettimeofday(&end_tv, NULL);
	sec = end_tv.tv_sec - start_tv.tv_sec;
	ms = end_tv.tv_usec - start_tv.tv_usec;

	diff = sec * 1000000 + ms;

	//printf("%10s:\t\t%fms\n", "Time elapsed", (double)((double)diff/1000.0));

	hipMemcpy(L1, d_L, ((numPic + 1)*numPic)/2, hipMemcpyDeviceToHost);
	hipFree(d_outputPixels);
	hipFree(d_L);
	hipDeviceReset();

	//for(i=0; i<((numPic + 1)*numPic)/2; i++)
	//{
	//	printf("%d:  %d\n", i, (unsigned int)L1[i]);
	//}

	int num = 0;
	for (int i=0;i<numPic; i++)
	{
		for(int j=i; j<numPic; j++)
		{
			//L[i*numPic+j] = L1[num];
			//L[j*numPic+i] = L1[num];
			num++;
		}
	}



	//memcpy(T, outputPixels, numPic * height * width);
	for(int round=0; round < numPic * height * width; round++)
	{
		T[round]=outputPixels[round];
	}


	//memcpy(T_test, &outputPixels[(numPic-1) * height * width], height * width);
	//free(outputPixels);
	//free(fdata);

	//matrix_reverse(T11,T,IMG_HEIGHT*IMG_WIDTH,TRAIN_NUM);


		fd = open("1.bmp", O_RDONLY);
		fstat(fd, &finfo);
		fdata = (char*) malloc(finfo.st_size);
		read (fd, fdata, finfo.st_size);
		//unsigned short *bitsperpixel = (unsigned short *)(&(fdata[BITS_PER_PIXEL_POS]));

		// ensure its 3 bytes per pixel
	
		data_pos = (unsigned short *)(&(fdata[IMG_DATA_OFFSET_POS]));
	
		//int imgdata_bytes = (int)finfo.st_size - (int)(*(data_pos));
		//printf("This file has %d bytes of image data, %d pixels\n", imgdata_bytes, imgdata_bytes / 3);

		width = *((int*)&fdata[18]);
		//printf("Width: %d\n", width);
		height = *((int*)&fdata[22]);
		//printf("Height: %d\n", height);

		fileSize = (int) finfo.st_size;	


	FILE *writeFile; 
	writeFile = fopen("round1.bmp","w+");
	for(ii = 0; ii < fileSize; ii++)
		if(ii<*data_pos)
		{
			fprintf(writeFile,"%c", fdata[ii]);
		} else {
			fprintf(writeFile,"%c", (unsigned char)T[ii - *data_pos]);
		}
	fclose(writeFile);


	//printf("%d %d %d %d %d %d %d %d %d %d \n",(unsigned int)T[0],(unsigned int)T[1],(int)T[36000-2],(int)T[36000-3],(int)T[36000-4], (int)T[36000-5],(int)T[36000-6],(int)T[36000-7],(int)T[36000-8], (int)T[36000-9]);



	//求T矩阵行的平均值  
    //calc_mean(T,m1);  
	//for(int i=0; i<36000;i++){
	//printf("m: %d: %d  %f\n", i, (unsigned int)m[i], m1[i]);
	//}

	for(int round=0; round < height * width; round++)
	{
		m1[round]= m[round];
	}


    calc_covariance_matrix(T,T1,L,m1); 
	free(T1);
	//printf("%qwqwqwqwqw: %d  %d\n", (unsigned int)L[0],(unsigned int)L[1]);

    iteration = 60;  
    cstrq(L,TRAIN_NUM,q,b,c);  
    flag = csstq(TRAIN_NUM,b,c,q,eps,iteration); 

	if (flag<0)  
    {  
        //printf("fucking failed!\n");  
    }else  
    {  
        //printf("success to get eigen value and vector\n");  
    }  

    num_q=0;  
    for (i=0;i<TRAIN_NUM;i++)  
    {  
        if (b[i]>1)  
        {  
            num_q++;  
        }  
    }  

    p_q = (double *)malloc(sizeof(double)*TRAIN_NUM*TRAIN_NUM); 


    projected_train = (double *)malloc(sizeof(double)*TRAIN_NUM*num_q);


    eigenvector = (double *)malloc(sizeof(double)*IMG_HEIGHT*IMG_WIDTH*num_q);


    pick_eignevalue(b,q,p_q,num_q);  
	//for(int round = 1; round <=TRAIN_NUM; round ++)
	//{
	//	printf("Eigenvalue: %e\n", b[round]);
	//}


    get_eigenface(p_q,T,num_q,projected_train,eigenvector); 
	

	fd1 = open(outputfname, O_RDONLY);
	

	fstat(fd1, &finfo);

	char *fdata1;
	fdata1 = (char*) malloc(finfo.st_size);


	read (fd1, fdata1, finfo.st_size);

	data_pos = (unsigned short *)(&(fdata1[IMG_DATA_OFFSET_POS]));

	
	
		width = *((int*)&fdata1[18]);
		//printf("Width: %d\n", width);
		height = *((int*)&fdata1[22]);
		//printf("Height: %d\n", height);

	char* pointer = &(fdata1[*data_pos]);

	unsigned char* d_test;
	hipMalloc((void**) &d_test,width * height * 3);
	hipMemcpy(d_test, pointer, width * height * 3, hipMemcpyHostToDevice);
	

	
	unsigned char* d_outtest;
	hipMalloc((void**) &d_outtest, width * height );
	hipMemset(d_outtest, 0, width * height);
	

	unsigned char* outtest = (unsigned char*) malloc(height * width );

	rgb2gray_test<<<grid, block>>>((uchar3*) d_test, (unsigned char*) d_outtest, 1);
	hipDeviceSynchronize();

	hipMemcpy(outtest, d_outtest, height * width, hipMemcpyDeviceToHost);


	memcpy(&(fdata1[*data_pos]), outtest,  height * width);
	FILE *writeFile1; 
	writeFile1 = fopen("round2.bmp","w+");
	for(ii = 0; ii < fileSize; ii++)
		fprintf(writeFile1,"%c", fdata1[ii]);
	fclose(writeFile1);

	for(int round=0; round < height * width; round++)
	{
		T_test[round]=outtest[round]-m[round];
	}

	projected_test = (double *)malloc(sizeof(double)*num_q*1);

    memset(projected_test,0,sizeof(double)*num_q);   
	matrix_mutil(projected_test,eigenvector,T_test,num_q,IMG_WIDTH*IMG_HEIGHT,1);  
 
    Euc_dist = (double *)malloc(sizeof(double)*TRAIN_NUM);  
    for (i=0;i<TRAIN_NUM;i++)  
    {  
        temp = 0;  
        for (j=0;j<num_q;j++)  
        {  
            temp = temp + (projected_test[j]-projected_train[j*TRAIN_NUM+i])*(projected_test[j]-projected_train[j*TRAIN_NUM+i]);  
        }  
        Euc_dist[i] = temp;  
		//printf("%d:  %e\n",i,  Euc_dist[i]);
    }  

    double min = Euc_dist[0];  
    int label;  
    for (i=0;i<TRAIN_NUM;i++)  
    {  
        if (min>=Euc_dist[i])  
        {  
            min = Euc_dist[i];  
            label = i;  
        }  
    }  
    printf("No.%d individual is mathcing!\n",label+1);  

	return 0;
} 




